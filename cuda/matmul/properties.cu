#include "hip/hip_runtime.h"
// nvcc ./properties.cu -o properties && ./properties 

#include <hip/hip_runtime.h>
#include <>
 
#include <stdlib.h>
#include <cstring>
#include <time.h>
#include <stdio.h>
#include <cmath>
#include <iostream>

 
int main()
{
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
 
    int devNo = 0;
    hipDeviceProp_t iProp;
    hipGetDeviceProperties(&iProp, devNo);
    printf("%32s  %d / %d\n", "device count", devNo, deviceCount);
    printf("%32s  %s\n", "device", iProp.name);
    printf("%32s  %d\n", "multiprocessor count", iProp.multiProcessorCount);
    printf("%32s  %d\n", "clock rate", iProp.clockRate);
    printf("%32s  %d.%d\n", "compute capability", iProp.major, iProp.minor);
    printf("%32s  %zd MiB\n", "total global memory", iProp.totalGlobalMem / 1024 / 1024);
    printf("%32s  %zd KiB\n", "total constant memory", iProp.totalConstMem / 1024);
    printf("%32s  %zd KiB\n", "shared memory per block", iProp.sharedMemPerBlock / 1024);
    printf("%32s  %d\n", "max threads per block", iProp.maxThreadsPerBlock);
    printf("%32s  [%d, %d, %d]\n", "max grid size", iProp.maxGridSize[0], iProp.maxGridSize[1], iProp.maxGridSize[2]);
    printf("%32s  [%d, %d, %d]\n", "max thread dims", iProp.maxThreadsDim[0], iProp.maxThreadsDim[1], iProp.maxThreadsDim[2]);
    printf("%32s  %d\n", "warp size", iProp.warpSize);
    printf("%32s  %d\n", "max threads per warp", iProp.maxThreadsPerBlock);
}
