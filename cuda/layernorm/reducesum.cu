// nvcc -o test ./reducesum.cu && ./test

#include <hip/hip_runtime.h>

#include <iostream>

// Kernel function to perform reduction on a 2D matrix along axis=1 (row-wise)
__global__ void reduceSumAxis1(float *input, float *output, int rows,
                               int cols) {
  extern __shared__ float sharedData[];

  int row = blockIdx.x;
  int tid = threadIdx.x;

  // Each thread loads one element from the row into shared memory
  sharedData[tid] = (tid < cols) ? input[row * cols + tid] : 0.0f;
  __syncthreads();

  // Perform reduction in shared memory
  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
      sharedData[tid] += sharedData[tid + s];
    }
    __syncthreads();
  }

  // Write the result of this row to the output array
  if (tid == 0) {
    output[row] = sharedData[0];
  }
}

// Host function to call the kernel and sum the rows
void sumMatrixAxis1(float *input, float *output, int rows, int cols) {
  float *d_input, *d_output;
  size_t inputSize = rows * cols * sizeof(float);
  size_t outputSize = rows * sizeof(float);
  size_t sharedMemSize = cols * sizeof(float);  // Each block sums one row

  // Allocate device memory
  hipMalloc((void **)&d_input, inputSize);
  hipMalloc((void **)&d_output, outputSize);

  // Copy data from host to device
  hipMemcpy(d_input, input, inputSize, hipMemcpyHostToDevice);

  // Launch the reduction kernel
  reduceSumAxis1<<<rows, cols, sharedMemSize>>>(d_input, d_output, rows, cols);

  // Copy the results back to the host
  hipMemcpy(output, d_output, outputSize, hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(d_input);
  hipFree(d_output);
}

int main() {
  int rows = 4;
  int cols = 4;
  int totalElements = rows * cols;
  float *h_input = (float *)malloc(totalElements * sizeof(float));
  float *h_output = (float *)malloc(rows * sizeof(float));

  // Initialize 2D matrix
  for (int i = 0; i < totalElements; ++i) {
    h_input[i] = static_cast<float>(i + 1);  // Fill with 1, 2, ..., n
  }

  // Calculate sum using CUDA
  sumMatrixAxis1(h_input, h_output, rows, cols);

  // Output the result
  std::cout << "Row-wise sum: ";
  for (int i = 0; i < rows; ++i) {
    std::cout << h_output[i] << " ";
  }
  std::cout << std::endl;

  // Free host memory
  free(h_input);
  free(h_output);

  return 0;
}